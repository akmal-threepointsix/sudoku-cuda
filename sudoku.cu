// CUDA runtime
#include <hip/hip_runtime.h>

// C++ libraries
#include <iostream>
#include <fstream>
#include <sstream>

using std::cout;
using std::cerr;
using std::endl;
using std::string;

// CUDA error checking. Source: https://github.com/NVIDIA/cuda-samples/blob/master/Common/helper_cuda.h
static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}
template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
// End

class Sudoku {
public:
	Sudoku() {
		memset(grid, 0, GRID_DIM * GRID_DIM);
	}

	void loadFromFile(std::ifstream& sudokuStream) {
		for (int row = 0; row < GRID_DIM; row++) {
			string rowString;
			std::getline(sudokuStream, rowString);
			for (int column = 0; column < GRID_DIM; column++) {
				grid[row * GRID_DIM + column] = rowString[column] - '0';
			}
		}
	}

	void print() {
		for (int i = 0; i < TOTAL_CELLS; i++) {
			if (i % 27 == 0) cout << "|" << endl << "|--------------------";
			if (i % 9 == 0) cout << "|" << endl;
			if (i % 3 == 0) cout << "|";
			if (grid[i]) {
				cout << (int)grid[i] << " ";
			}
			else {
				cout << ". ";
			}
		}
		cout << "|" << endl;
	}

	static const int GRID_DIM = 9;
	static const int BOX_DIM = 3;
	static const int TOTAL_CELLS = GRID_DIM * GRID_DIM;
	unsigned char grid[GRID_DIM * GRID_DIM];
};

enum class SudokuStatus {
	Added, // Added some digits to the sudoku
	NothingAdded, // Didn't add any digit (but there are valid options)
	Failed, // Current sudoku cannot be solved because of some incorrect guess
	Solved // Sudoku is completely and correctly solved
};

// Structure of Arrays
struct UsedDigitsBitmasks {
	uint16_t rowContains[9];
	uint16_t columnContains[9];
	uint16_t boxContains[9];
};

// Find used digits in rows, columns and boxes. Save used digits in appropriate bitmasks
// Side effects: Set sudokuStatus=failed, if some row, column or box contains two same digits
__device__ void findUsedDigits(UsedDigitsBitmasks* sm_usedDigitsBitmasks, uint8_t* currentBlockGrid, SudokuStatus& sudokuStatus) {
	if (threadIdx.x < Sudoku::GRID_DIM) { // Rows
		sm_usedDigitsBitmasks->rowContains[threadIdx.x] = 0;
		for (uint8_t row = 0; row < Sudoku::GRID_DIM; row++) {
			int digit = currentBlockGrid[threadIdx.x * Sudoku::GRID_DIM + row];
			if (digit) {
				if (sm_usedDigitsBitmasks->rowContains[threadIdx.x] >> (digit - 1) & 1) { // Two same digits in one row
					sudokuStatus = SudokuStatus::Failed;
				}
				sm_usedDigitsBitmasks->rowContains[threadIdx.x] |= 1 << (digit - 1);
			}
		}
	}
	else if (Sudoku::GRID_DIM <= threadIdx.x && threadIdx.x < Sudoku::GRID_DIM * 2) { // Columns
		const uint8_t orderIdx = threadIdx.x - Sudoku::GRID_DIM; // [0, 8]
		sm_usedDigitsBitmasks->columnContains[orderIdx] = 0;
		for (uint8_t column = 0; column < Sudoku::GRID_DIM; column++) {
			uint8_t digit = currentBlockGrid[column * Sudoku::GRID_DIM + orderIdx];
			if (digit) {
				if (sm_usedDigitsBitmasks->columnContains[orderIdx] >> (digit - 1) & 1) { // Two same digits in one column
					sudokuStatus = SudokuStatus::Failed;
				}
				sm_usedDigitsBitmasks->columnContains[orderIdx] |= 1 << (digit - 1);
			}
		}
	}
	else if (Sudoku::GRID_DIM * 2 <= threadIdx.x && threadIdx.x < Sudoku::GRID_DIM * 3) { // Boxes
		const uint8_t orderIdx = threadIdx.x - Sudoku::GRID_DIM * 2; // [0, 8]
		sm_usedDigitsBitmasks->boxContains[orderIdx] = 0;
		for (uint8_t row = (orderIdx / 3) * 3; row < ((orderIdx / 3 + 1) * 3); row++) {
			for (uint8_t column = (orderIdx % 3) * 3; column < ((orderIdx % 3 + 1) * 3); column++) {
				uint8_t digit = currentBlockGrid[row * Sudoku::GRID_DIM + column];
				if (digit) {
					if ((sm_usedDigitsBitmasks->boxContains[orderIdx] >> (digit - 1)) & 1) { // Two same digits in one box
						sudokuStatus = SudokuStatus::Failed;
					}
					sm_usedDigitsBitmasks->boxContains[orderIdx] |= 1 << (digit - 1);
				}
			}
		}
	}
}

__device__ void tryAddDigit(uint16_t& allPossibleDigits, UsedDigitsBitmasks* sm_usedDigitsBitmasks, uint8_t* currentBlockGrid, SudokuStatus& sm_sudokuStatus) {
	sm_sudokuStatus = SudokuStatus::NothingAdded;

	uint8_t row = threadIdx.x / Sudoku::GRID_DIM;
	uint8_t column = threadIdx.x % Sudoku::GRID_DIM;
	allPossibleDigits = (sm_usedDigitsBitmasks->rowContains[row]
		| sm_usedDigitsBitmasks->columnContains[column]
		| sm_usedDigitsBitmasks->boxContains[(row / 3) * 3 + (column / 3)]);

	uint16_t unusedDigit = 0;
	for (uint8_t possibleDigit = 0; possibleDigit < Sudoku::GRID_DIM; possibleDigit++) {
		const bool isUnused = (allPossibleDigits & (1 << possibleDigit)) == 0;
		if (isUnused) {
			if (unusedDigit != 0) { // More than one unused digit
				unusedDigit = 10;
				break;
			}
			else {
				unusedDigit = possibleDigit + 1;
			}
		}
	}
	if (unusedDigit == 0) { // Could not find any unused digit
		sm_sudokuStatus = SudokuStatus::Failed;
	}
	else if (unusedDigit <= 9) { // Found exactly one unused digit
		currentBlockGrid[threadIdx.x] = unusedDigit;
		sm_sudokuStatus = SudokuStatus::Added;
	}
}

// Idea source: https://github.com/evcu/cuda-sudoku-solver
// Fill cells that can be filled by only one digit
// Side effects: If there are no such cells, then find cells with the minimum number of possible digits, 
// create new sudokus for each possible digit and repeat the process for new sudokus
__global__ void fillSudoku(unsigned char* gm_allSudokuGrids, unsigned char* gm_solvedSudoku, uint32_t* gm_isBlockActive, bool* gm_isSolved) {
	uint8_t* currentBlockGrid = gm_allSudokuGrids + (Sudoku::TOTAL_CELLS * blockIdx.x); // Get appropriate grid with pointer arithmetics

	const bool isBlockActive = gm_isBlockActive[blockIdx.x] == 1;
	const bool isThreadActive = threadIdx.x < 81; // 81 threads for 81 cells

	__shared__ SudokuStatus sm_sudokuStatus;
	__shared__ UsedDigitsBitmasks sm_usedDigitsBitmasks; // Bitmasks for used digits
	__shared__ int sm_minPossibleDigits;
	__shared__ int sm_schedulingThread;

	if (isBlockActive && isThreadActive) {
		const bool isFirstThread = threadIdx.x == 0;
		if (isFirstThread) {
			sm_sudokuStatus = SudokuStatus::Added;
		}
		__syncthreads();

		uint16_t allPossibleDigits;
		while (sm_sudokuStatus == SudokuStatus::Added) { // This loop fills cells that can be filled with only one digit
			findUsedDigits(&sm_usedDigitsBitmasks, currentBlockGrid, sm_sudokuStatus);
			__syncthreads();

			if (sm_sudokuStatus != SudokuStatus::Failed) {
				if (isFirstThread) {
					sm_sudokuStatus = SudokuStatus::Solved; // Assume that the sudoku is solved
				}
				__syncthreads();
				allPossibleDigits = 0;
				if (currentBlockGrid[threadIdx.x] == 0) {
					tryAddDigit(allPossibleDigits, &sm_usedDigitsBitmasks, currentBlockGrid, sm_sudokuStatus);
				}
			}
			__syncthreads();
		}

		const bool isSolved = sm_sudokuStatus == SudokuStatus::Solved;
		const bool isFailed = sm_sudokuStatus == SudokuStatus::Failed;
		const bool cannotAdd = sm_sudokuStatus == SudokuStatus::NothingAdded;

		if (isSolved && isFirstThread) { // Success. Sudoku was completely and correctly solved
			memcpy(gm_solvedSudoku, currentBlockGrid, Sudoku::TOTAL_CELLS);
			*gm_isSolved = true;
		}
		else if (isFailed && isFirstThread) { // Failed. Stop solving this particular sudoku
			gm_isBlockActive[blockIdx.x] = 0;
		}
		else if (cannotAdd) { // Fork
			if (isFirstThread) {
				sm_minPossibleDigits = 9;
				sm_schedulingThread = blockDim.x;
			}
			__syncthreads();

			uint8_t possibleDigitsCount = 0;
			if (allPossibleDigits != 0) {
				for (int possibleDigit = 0; possibleDigit < 9; possibleDigit++) {
					if ((allPossibleDigits & (1 << possibleDigit)) == 0) {
						possibleDigitsCount++;
					}
				}
				atomicMin(&sm_minPossibleDigits, possibleDigitsCount);
			}
			__syncthreads();

			if (possibleDigitsCount == sm_minPossibleDigits) {
				atomicMin(&sm_schedulingThread, threadIdx.x);
			}
			__syncthreads();

			if (sm_schedulingThread == threadIdx.x) {
				//Find a suitable block to schedule the fork for each extra value.
				for (int i = 0, k = 1; i < 9; i++) {
					if ((allPossibleDigits & (1 << i)) == 0) {
						if (k == 1) {
							// first possibility stays with the current block
							currentBlockGrid[threadIdx.x] = i + 1;
						}
						else {
							// look for suitable block
							for (int j = 0; j < gridDim.x; j++) {
								atomicCAS(gm_isBlockActive + j, 0, gridDim.x * blockIdx.x + threadIdx.x + 2);
								if (gm_isBlockActive[j] == (gridDim.x * blockIdx.x + threadIdx.x + 2)) {
									memcpy(gm_allSudokuGrids + j * 81, currentBlockGrid, Sudoku::TOTAL_CELLS);
									gm_allSudokuGrids[j * 81 + threadIdx.x] = i + 1;
									gm_isBlockActive[j] = 1;
									break;
								}
							}
						}
						k++;
					}
				}
			}
			__syncthreads();
		}
	}
}

void SolveSudoku(Sudoku* sudoku) {
	// Initialize variables
	const int h_threadsNum = 96; // Must be multiple of 32 and greater than 81
	const int h_blocksNum = 20000; // Some random big number

	unsigned char* d_allSudokuGrids;
	checkCudaErrors(hipMalloc(&d_allSudokuGrids, Sudoku::TOTAL_CELLS * h_blocksNum)); // All grids are stored here
	checkCudaErrors(hipMemcpy(d_allSudokuGrids, (*sudoku).grid, Sudoku::TOTAL_CELLS, hipMemcpyHostToDevice));

	unsigned char* d_solvedSudoku;
	checkCudaErrors(hipMalloc(&d_solvedSudoku, Sudoku::TOTAL_CELLS));

	uint32_t* d_isBlockActive;
	checkCudaErrors(hipMalloc(&d_isBlockActive, h_blocksNum * sizeof(uint32_t)));
	checkCudaErrors(hipMemset(d_isBlockActive, 0, h_blocksNum * sizeof(uint32_t)));
	checkCudaErrors(hipMemset(d_isBlockActive, 1, 1)); // d_isBlockActive[0] = true;

	bool* d_isSolved;
	checkCudaErrors(hipMalloc(&d_isSolved, 1));
	checkCudaErrors(hipMemset(d_isSolved, false, 1)); // *d_isSolved = false;

	bool h_isSolved = false;
	// Solving sudoku
	while (!h_isSolved) { // Limit it by 100 iterations, so it never gets stuck?
		fillSudoku<<<h_blocksNum, h_threadsNum>>>(d_allSudokuGrids, d_solvedSudoku, d_isBlockActive, d_isSolved);
		checkCudaErrors(hipDeviceSynchronize()); // is it necessary?
		checkCudaErrors(hipMemcpy(&h_isSolved, d_isSolved, 1, hipMemcpyDeviceToHost));
	}

	// Copying solved sudoku to host
	checkCudaErrors(hipMemcpy((*sudoku).grid, d_solvedSudoku, Sudoku::TOTAL_CELLS, hipMemcpyDeviceToHost));

	// Cleanup
	checkCudaErrors(hipFree(d_allSudokuGrids));
	checkCudaErrors(hipFree(d_solvedSudoku));
	checkCudaErrors(hipFree(d_isBlockActive));
	checkCudaErrors(hipFree(d_isSolved));
}

int main(int argc, char** argv) {
	// Usage
	if (argc < 2) {
		cerr << "Usage: sudoku filename" << endl;
		return -1;
	}

	// Load Sudoku from file
	std::ifstream sudokuFile(argv[1]);
	if (sudokuFile.fail()) {
		cout << argv[1] << " not found!" << endl;
		return false;
	}

	int sudokuCounter = 0;
	double totalTimeSec = 0;
	clock_t startTime, endTime;
	string newline;
	do {
		Sudoku h_sudoku;
		h_sudoku.loadFromFile(sudokuFile);
		// Print unsolved sudoku
		h_sudoku.print();

		startTime = clock();
		SolveSudoku(&h_sudoku);
		endTime = clock();

		// Print solved sudoku
		h_sudoku.print();

		totalTimeSec += (double(endTime - startTime) / CLOCKS_PER_SEC);
		sudokuCounter++;
	} while (std::getline(sudokuFile, newline));
	cout << endl;
	cout << "Solved " << sudokuCounter << " sudokus in " << totalTimeSec << " seconds!" << endl;

	return 0;
}